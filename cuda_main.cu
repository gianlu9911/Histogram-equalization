#include "UtilityCuda.cu"

void equalizeImageWithCUDA(const cv::Mat& inputImage)
{
    int width = inputImage.cols;
    int height = inputImage.rows;

    // Allocate device memory
    unsigned char* d_image;
    unsigned char* d_output;
    int *d_hist_r, *d_hist_g, *d_hist_b;
    unsigned char *d_cdf_r, *d_cdf_g, *d_cdf_b;

    // Timing variables
    hipEvent_t start, stop, histStart, histStop, cdfStart, cdfStop, eqStart, eqStop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventCreate(&histStart));
    CUDA_CHECK(hipEventCreate(&histStop));
    CUDA_CHECK(hipEventCreate(&cdfStart));
    CUDA_CHECK(hipEventCreate(&cdfStop));
    CUDA_CHECK(hipEventCreate(&eqStart));
    CUDA_CHECK(hipEventCreate(&eqStop));

    // Start timing for the entire process
    CUDA_CHECK(hipEventRecord(start));

    // Flatten the input image and transfer to device memory
    CUDA_CHECK(hipMalloc(&d_image, width * height * 3 * sizeof(unsigned char)));
    CUDA_CHECK(hipMemcpy(d_image, inputImage.data, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_output, width * height * 3 * sizeof(unsigned char)));

    // Allocate device memory for histograms
    CUDA_CHECK(hipMalloc(&d_hist_r, 256 * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_hist_g, 256 * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_hist_b, 256 * sizeof(int)));

    CUDA_CHECK(hipMemset(d_hist_r, 0, 256 * sizeof(int)));
    CUDA_CHECK(hipMemset(d_hist_g, 0, 256 * sizeof(int)));
    CUDA_CHECK(hipMemset(d_hist_b, 0, 256 * sizeof(int)));

    // Start timing for histogram computation
    CUDA_CHECK(hipEventRecord(histStart));

    // Compute histogram on the device
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    computeHistogram<<<grid, block>>>(d_image, d_hist_r, d_hist_g, d_hist_b, width, height);

    // Stop timing for histogram computation
    CUDA_CHECK(hipEventRecord(histStop));
    CUDA_CHECK(hipEventSynchronize(histStop));

    float histTime = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&histTime, histStart, histStop));
    std::cout << "Histogram computation time: " << histTime << " ms" << std::endl;

    // Copy the histograms from device to host
    int h_hist_r[256] = {0};
    int h_hist_g[256] = {0};
    int h_hist_b[256] = {0};

    CUDA_CHECK(hipMemcpy(h_hist_r, d_hist_r, 256 * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_hist_g, d_hist_g, 256 * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_hist_b, d_hist_b, 256 * sizeof(int), hipMemcpyDeviceToHost));

    // Normalize the histograms
    int max_height = 256; // Height for displaying the histogram images
    normalizeHistogram(h_hist_r, 256, max_height);
    normalizeHistogram(h_hist_g, 256, max_height);
    normalizeHistogram(h_hist_b, 256, max_height);

    // Create black images for plotting the histograms
    cv::Mat hist_img_r = cv::Mat::zeros(cv::Size(256, max_height), CV_8UC3);
    cv::Mat hist_img_g = cv::Mat::zeros(cv::Size(256, max_height), CV_8UC3);
    cv::Mat hist_img_b = cv::Mat::zeros(cv::Size(256, max_height), CV_8UC3);

    // Draw the histograms on the black images
    //drawHistogram(h_hist_r, 256, hist_img_r, cv::Scalar(0, 0, 255)); // Red histogram in red color
    //drawHistogram(h_hist_g, 256, hist_img_g, cv::Scalar(0, 255, 0)); // Green histogram in green color
    //drawHistogram(h_hist_b, 256, hist_img_b, cv::Scalar(255, 0, 0)); // Blue histogram in blue color

    // Combine the individual histograms into one image
    cv::Mat combined_hist;
    cv::hconcat(hist_img_r, hist_img_g, combined_hist);
    cv::hconcat(combined_hist, hist_img_b, combined_hist);

    // Display the combined histogram image
    //cv::imshow("RGB Histograms", combined_hist);
    //cv::waitKey(0);

    // Allocate memory for CDFs
    CUDA_CHECK(hipMalloc(&d_cdf_r, 256 * sizeof(unsigned char)));
    CUDA_CHECK(hipMalloc(&d_cdf_g, 256 * sizeof(unsigned char)));
    CUDA_CHECK(hipMalloc(&d_cdf_b, 256 * sizeof(unsigned char)));

    // Start timing for CDF computation
    CUDA_CHECK(hipEventRecord(cdfStart));

    // Compute CDFs on the device
    computeCDF<<<1, 256>>>(d_hist_r, d_cdf_r, width, height);
    computeCDF<<<1, 256>>>(d_hist_g, d_cdf_g, width, height);
    computeCDF<<<1, 256>>>(d_hist_b, d_cdf_b, width, height);

    // Stop timing for CDF computation
    CUDA_CHECK(hipEventRecord(cdfStop));
    CUDA_CHECK(hipEventSynchronize(cdfStop));

    float cdfTime = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&cdfTime, cdfStart, cdfStop));
    std::cout << "CDF computation time: " << cdfTime << " ms" << std::endl;

    // Launch the kernel to apply histogram equalization
    CUDA_CHECK(hipEventRecord(eqStart));
    equalizeRGBImageTiled<<<grid, block>>>(d_image, d_output, width, height, d_cdf_r, d_cdf_g, d_cdf_b);
    CUDA_CHECK(hipEventRecord(eqStop));
    CUDA_CHECK(hipEventSynchronize(eqStop));

    // Stop timing for histogram equalization
    float eqTime = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&eqTime, eqStart, eqStop));
    std::cout << "Histogram equalization time: " << eqTime << " ms" << std::endl;

    // Copy result back to host
    cv::Mat outputImage(height, width, CV_8UC3);
    CUDA_CHECK(hipMemcpy(outputImage.data, d_output, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));

    // Cleanup
    CUDA_CHECK(hipFree(d_image));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_hist_r));
    CUDA_CHECK(hipFree(d_hist_g));
    CUDA_CHECK(hipFree(d_hist_b));
    CUDA_CHECK(hipFree(d_cdf_r));
    CUDA_CHECK(hipFree(d_cdf_g));
    CUDA_CHECK(hipFree(d_cdf_b));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUDA_CHECK(hipEventDestroy(histStart));
    CUDA_CHECK(hipEventDestroy(histStop));
    CUDA_CHECK(hipEventDestroy(cdfStart));
    CUDA_CHECK(hipEventDestroy(cdfStop));
    CUDA_CHECK(hipEventDestroy(eqStart));
    CUDA_CHECK(hipEventDestroy(eqStop));

    // Save the processed image
    cv::imwrite("outputs/my_image_cuda.jpg", outputImage);
}

int main()
{
    // Load an image with OpenCV
    cv::Mat inputImage = cv::imread("images/f.jpg", cv::IMREAD_COLOR); // Load in color (3 channels)
    if (inputImage.empty()) {
        std::cerr << "Error: Could not load image!" << std::endl;
        return -1;
    }

    // Equalize the image using CUDA
    equalizeImageWithCUDA(inputImage);

    return 0;
}
