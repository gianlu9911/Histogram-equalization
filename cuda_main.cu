#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

#define HISTOGRAM_SIZE 256
#define TILE_SIZE 32

__global__ void calculate_histogram_tiled(unsigned char* d_channel, int* d_histogram, int rows, int cols) {
    __shared__ int local_histogram[HISTOGRAM_SIZE];

    int thread_id = threadIdx.y * blockDim.x + threadIdx.x;
    int global_x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int global_y = blockIdx.y * TILE_SIZE + threadIdx.y;

    if (thread_id < HISTOGRAM_SIZE) local_histogram[thread_id] = 0;
    __syncthreads();

    if (global_x < cols && global_y < rows) {
        atomicAdd(&local_histogram[d_channel[global_y * cols + global_x]], 1);
    }
    __syncthreads();

    if (thread_id < HISTOGRAM_SIZE) {
        atomicAdd(&d_histogram[thread_id], local_histogram[thread_id]);
    }
}

__global__ void prefix_sum_kernel(int* d_histogram, int* d_cdf) {
    __shared__ int temp[HISTOGRAM_SIZE];

    int idx = threadIdx.x;
    if (idx < HISTOGRAM_SIZE) temp[idx] = d_histogram[idx];
    __syncthreads();

    for (int offset = 1; offset < HISTOGRAM_SIZE; offset *= 2) {
        int val = 0;
        if (idx >= offset) val = temp[idx - offset];
        __syncthreads();
        temp[idx] += val;
        __syncthreads();
    }

    if (idx < HISTOGRAM_SIZE) d_cdf[idx] = temp[idx];
}

__global__ void histogram_equalization_tiled(unsigned char* d_channel, int* d_cdf, int rows, int cols, int cdf_min, int cdf_max) {
    int x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int y = blockIdx.y * TILE_SIZE + threadIdx.y;

    if (x < cols && y < rows) {
        int pixel = d_channel[y * cols + x];
        d_channel[y * cols + x] = (unsigned char)(((d_cdf[pixel] - cdf_min) * 255) / (cdf_max - cdf_min));
    }
}

void histogram_equalization_rgb_cuda(cv::Mat& img) {
    int rows = img.rows, cols = img.cols;
    int img_size = rows * cols;
    int channel_size = img_size * sizeof(unsigned char);

    std::vector<cv::Mat> channels(3);
    cv::split(img, channels);  

    unsigned char *h_r, *h_g, *h_b;
    hipHostAlloc((void**)&h_r, channel_size, hipHostMallocDefault);
    hipHostAlloc((void**)&h_g, channel_size, hipHostMallocDefault);
    hipHostAlloc((void**)&h_b, channel_size, hipHostMallocDefault);

    memcpy(h_r, channels[2].data, channel_size);  // R
    memcpy(h_g, channels[1].data, channel_size);  // G
    memcpy(h_b, channels[0].data, channel_size);  // B

    unsigned char *d_r, *d_g, *d_b;
    int *d_histogram_r, *d_histogram_g, *d_histogram_b;
    int *d_cdf_r, *d_cdf_g, *d_cdf_b;

    hipMalloc(&d_r, channel_size);
    hipMalloc(&d_g, channel_size);
    hipMalloc(&d_b, channel_size);
    hipMalloc(&d_histogram_r, HISTOGRAM_SIZE * sizeof(int));
    hipMalloc(&d_histogram_g, HISTOGRAM_SIZE * sizeof(int));
    hipMalloc(&d_histogram_b, HISTOGRAM_SIZE * sizeof(int));
    hipMalloc(&d_cdf_r, HISTOGRAM_SIZE * sizeof(int));
    hipMalloc(&d_cdf_g, HISTOGRAM_SIZE * sizeof(int));
    hipMalloc(&d_cdf_b, HISTOGRAM_SIZE * sizeof(int));

    hipMemcpy(d_r, h_r, channel_size, hipMemcpyHostToDevice);
    hipMemcpy(d_g, h_g, channel_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, channel_size, hipMemcpyHostToDevice);

    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid((cols + TILE_SIZE - 1) / TILE_SIZE, (rows + TILE_SIZE - 1) / TILE_SIZE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float total_time = 0, histogram_time = 0, cdf_time = 0, equalization_time = 0, memory_transfer_time = 0;

    hipEventRecord(start);
    
    hipEvent_t hist_start, hist_stop;
    hipEventCreate(&hist_start);
    hipEventCreate(&hist_stop);
    hipEventRecord(hist_start);

    calculate_histogram_tiled<<<grid, block>>>(d_r, d_histogram_r, rows, cols);
    calculate_histogram_tiled<<<grid, block>>>(d_g, d_histogram_g, rows, cols);
    calculate_histogram_tiled<<<grid, block>>>(d_b, d_histogram_b, rows, cols);

    hipEventRecord(hist_stop);
    hipEventSynchronize(hist_stop);
    hipEventElapsedTime(&histogram_time, hist_start, hist_stop);

    hipEvent_t cdf_start, cdf_stop;
    hipEventCreate(&cdf_start);
    hipEventCreate(&cdf_stop);
    hipEventRecord(cdf_start);

    prefix_sum_kernel<<<1, HISTOGRAM_SIZE>>>(d_histogram_r, d_cdf_r);
    prefix_sum_kernel<<<1, HISTOGRAM_SIZE>>>(d_histogram_g, d_cdf_g);
    prefix_sum_kernel<<<1, HISTOGRAM_SIZE>>>(d_histogram_b, d_cdf_b);

    hipEventRecord(cdf_stop);
    hipEventSynchronize(cdf_stop);
    hipEventElapsedTime(&cdf_time, cdf_start, cdf_stop);

    int h_cdf_r[HISTOGRAM_SIZE], h_cdf_g[HISTOGRAM_SIZE], h_cdf_b[HISTOGRAM_SIZE];
    hipMemcpy(h_cdf_r, d_cdf_r, HISTOGRAM_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_cdf_g, d_cdf_g, HISTOGRAM_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_cdf_b, d_cdf_b, HISTOGRAM_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    int cdf_min_r = h_cdf_r[0], cdf_max_r = h_cdf_r[255];
    int cdf_min_g = h_cdf_g[0], cdf_max_g = h_cdf_g[255];
    int cdf_min_b = h_cdf_b[0], cdf_max_b = h_cdf_b[255];

    hipEvent_t equal_start, equal_stop;
    hipEventCreate(&equal_start);
    hipEventCreate(&equal_stop);
    hipEventRecord(equal_start);

    histogram_equalization_tiled<<<grid, block>>>(d_r, d_cdf_r, rows, cols, cdf_min_r, cdf_max_r);
    histogram_equalization_tiled<<<grid, block>>>(d_g, d_cdf_g, rows, cols, cdf_min_g, cdf_max_g);
    histogram_equalization_tiled<<<grid, block>>>(d_b, d_cdf_b, rows, cols, cdf_min_b, cdf_max_b);

    hipEventRecord(equal_stop);
    hipEventSynchronize(equal_stop);
    hipEventElapsedTime(&equalization_time, equal_start, equal_stop);

    hipEvent_t mem_start, mem_stop;
    hipEventCreate(&mem_start);
    hipEventCreate(&mem_stop);
    hipEventRecord(mem_start);

    hipMemcpy(h_r, d_r, channel_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_g, d_g, channel_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_b, d_b, channel_size, hipMemcpyDeviceToHost);

    hipEventRecord(mem_stop);
    hipEventSynchronize(mem_stop);
    hipEventElapsedTime(&memory_transfer_time, mem_start, mem_stop);

    memcpy(channels[2].data, h_r, channel_size);
    memcpy(channels[1].data, h_g, channel_size);
    memcpy(channels[0].data, h_b, channel_size);
    cv::merge(channels, img);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&total_time, start, stop);

    std::cout << "Total execution time: " << total_time << " ms" << std::endl;
    std::cout << "Histogram computation time: " << histogram_time << " ms" << std::endl;
    std::cout << "CDF computation time: " << cdf_time << " ms" << std::endl;
    std::cout << "Equalization computation time: " << equalization_time << " ms" << std::endl;
    std::cout << "Memory transfer time: " << memory_transfer_time << " ms" << std::endl;

    hipHostFree(h_r);
    hipHostFree(h_g);
    hipHostFree(h_b);
    hipFree(d_r);
    hipFree(d_g);
    hipFree(d_b);
    hipFree(d_histogram_r);
    hipFree(d_histogram_g);
    hipFree(d_histogram_b);
    hipFree(d_cdf_r);
    hipFree(d_cdf_g);
    hipFree(d_cdf_b);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(hist_start);
    hipEventDestroy(hist_stop);
    hipEventDestroy(cdf_start);
    hipEventDestroy(cdf_stop);
    hipEventDestroy(equal_start);
    hipEventDestroy(equal_stop);
    hipEventDestroy(mem_start);
    hipEventDestroy(mem_stop);
}

int main() {
    cv::Mat img = cv::imread("images/img4.bmp");
    if (img.empty()) {
        std::cerr << "Error loading image!" << std::endl;
        return -1;
    }

    cv::imshow("Original Image", img);
    histogram_equalization_rgb_cuda(img);
    
    cv::imshow("Equalized Image", img);
    cv::waitKey(0);  // Wait indefinitely for a key press
    cv::destroyAllWindows();  // Close all OpenCV windows

    return 0;
}