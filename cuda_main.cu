#include "UtilityCuda.cu"

int main() {
    std::string inputPath = "../images/img1.bmp";

    cv::Mat img = cv::imread(inputPath, cv::IMREAD_GRAYSCALE);
    if (img.empty()) {
        std::cerr << "Error: Could not load image at " << inputPath << std::endl;
        return -1;
    }

    std::string csvPath = "../execution_times.csv";

    std::ofstream csvFile(csvPath, std::ios::out);

    // If empty write the head of the csv
    if (csvFile.tellp() == 0) {
        csvFile << "Image Size,Histogram Time (ms),CDF Time (ms),Equalization Time (ms),Total Time (ms),Threads,Blocks\n";
    }

    std::vector<int> sizes = {128, 256, 512, 1024, 2048}; // size test

    for (int size : sizes) {
        cv::Mat resized_img;
        cv::resize(img, resized_img, cv::Size(size, size));

        int width = resized_img.cols;
        int height = resized_img.rows;

        unsigned char *d_input, *d_output;
        int *d_hist, *d_cdf;
        CUDA_CHECK(hipMalloc(&d_input, width * height * sizeof(unsigned char)));
        CUDA_CHECK(hipMalloc(&d_output, width * height * sizeof(unsigned char)));
        CUDA_CHECK(hipMalloc(&d_hist, 256 * sizeof(int)));
        CUDA_CHECK(hipMalloc(&d_cdf, 256 * sizeof(int)));

        CUDA_CHECK(hipMemcpy(d_input, resized_img.data, width * height * sizeof(unsigned char), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemset(d_hist, 0, 256 * sizeof(int)));

        hipEvent_t start, stop;
        float milliseconds = 0.0f;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));

        int threads = 256;
        int blocks_histogram = (width * height + threads - 1) / threads;

        // Histogram computation
        CUDA_CHECK(hipEventRecord(start));
        compute_histogram<<<blocks_histogram, threads>>>(d_input, d_hist, width, height);
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        float histogram_time = milliseconds;

        // CDF computation
        CUDA_CHECK(hipEventRecord(start));
        compute_cdf<<<1, 256>>>(d_hist, d_cdf, width * height);
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        float cdf_time = milliseconds;

        // Equalize image
        CUDA_CHECK(hipEventRecord(start));
        equalize_image<<<blocks_histogram, threads>>>(d_output, d_input, d_cdf, width, height);
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        float equalization_time = milliseconds;

        float total_time = histogram_time + cdf_time + equalization_time;

        std::cout << "Image Size: " << size << "x" << size 
                  << ", Histogram Time (ms): " << histogram_time 
                  << ", CDF Time (ms): " << cdf_time 
                  << ", Equalization Time (ms): " << equalization_time 
                  << ", Total Time (ms): " << total_time << std::endl;

        // Save results 
        csvFile << size << "x" << size << ","
                << histogram_time << ","
                << cdf_time << ","
                << equalization_time << ","
                << total_time << ","
                << threads << ","
                << blocks_histogram << "\n";

        
        CUDA_CHECK(hipFree(d_input));
        CUDA_CHECK(hipFree(d_output));
        CUDA_CHECK(hipFree(d_hist));
        CUDA_CHECK(hipFree(d_cdf));

        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));
    }

    csvFile.close(); 

    std::cout << "Execution times saved to " << csvPath << std::endl;
    return 0;
}