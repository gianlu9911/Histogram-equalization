#include "hip/hip_runtime.h"
#include "UtilityCuda.cu"


#include <fstream>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ << ": " \
                  << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

__global__ void warmUpKernel() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // No operation, just to warm up the device
    if (idx == 0) {
        printf("CUDA device warmed up!\n");
    }
}


void equalizeImageWithCUDAGrayscale(const cv::Mat& inputImage) {
    int width = inputImage.cols;
    int height = inputImage.rows;
    const int TILE_WIDTH = 16;  // Tile width for shared memory
    const int TILE_HEIGHT = 16; // Tile height for shared memory

    // Device memory allocation
    unsigned char *d_image, *d_output;
    int *d_hist;
    unsigned char *d_cdf;

    // CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Start overall time measurement
    CUDA_CHECK(hipEventRecord(start));

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_image, width * height * sizeof(unsigned char)));
    CUDA_CHECK(hipMemcpy(d_image, inputImage.data, width * height * sizeof(unsigned char), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_output, width * height * sizeof(unsigned char)));
    CUDA_CHECK(hipMalloc(&d_hist, 256 * sizeof(int)));
    CUDA_CHECK(hipMemset(d_hist, 0, 256 * sizeof(int)));

    CUDA_CHECK(hipMalloc(&d_cdf, 256 * sizeof(unsigned char)));

    // Warm-up kernel
    warmUpKernel<<<1, 1>>>();
    CUDA_CHECK(hipDeviceSynchronize());

    // Histogram computation
    dim3 block(TILE_WIDTH, TILE_HEIGHT);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    computeHistogramGrayscale<<<grid, block>>>(d_image, d_hist, width, height);
    CUDA_CHECK(hipDeviceSynchronize());

    // CDF computation
    computeCDFGrayscale<<<1, 256>>>(d_hist, d_cdf, width, height);
    CUDA_CHECK(hipDeviceSynchronize());

    // Equalization
    equalizeGrayscaleImage<<<grid, block>>>(d_image, d_output, width, height, d_cdf);
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    cv::Mat outputImage(height, width, CV_8UC1);
    CUDA_CHECK(hipMemcpy(outputImage.data, d_output, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost));

    // Cleanup
    CUDA_CHECK(hipFree(d_image));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_hist));
    CUDA_CHECK(hipFree(d_cdf));

    // Save the output
    cv::imwrite("../outputs/cuda_equalized_grayscale_image_" + std::to_string(width) + "_" + std::to_string(height) + ".jpg", outputImage);

    // Stop timing
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    // Calculate the total elapsed time
    float totalMilliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&totalMilliseconds, start, stop));
    std::cout << "Total Histogram Equalization (Grayscale) executed in " << totalMilliseconds << " ms.\n";

    // Write execution times to CSV file
    std::ofstream csvFile("../execution_times_cuda.csv", std::ios::app);
    if (csvFile.is_open()) {
        csvFile << "Total execution," 
                << totalMilliseconds << ","
                << width << ","
                << height << ","
                << 1 << ","  // Grayscale image has 1 channel
                << TILE_WIDTH << ","
                << TILE_HEIGHT << ","
                << block.x << ","
                << block.y << "\n";
        csvFile.close();
    } else {
        std::cerr << "Error: Could not open CSV file for logging." << std::endl;
    }
}


void equalizeImageWithCUDA(const cv::Mat& inputImage, int tile_width, int tile_height) {
    int width = inputImage.cols;
    int height = inputImage.rows;
    int channels = inputImage.channels();  // Get the number of channels

    // Allocate memory for input image and output image
    unsigned char *d_image, *d_output;
    int *d_hist_r, *d_hist_g, *d_hist_b;
    unsigned char *d_cdf_r, *d_cdf_g, *d_cdf_b;

    // Allocate device memory (ensure it's the correct size)
    size_t imageSize = width * height * 3 * sizeof(unsigned char);
    size_t histSize = 256 * sizeof(int);

    hipMalloc(&d_image, imageSize);
    hipMalloc(&d_output, imageSize);
    hipMalloc(&d_hist_r, histSize);
    hipMalloc(&d_hist_g, histSize);
    hipMalloc(&d_hist_b, histSize);
    hipMalloc(&d_cdf_r, 256 * sizeof(unsigned char));
    hipMalloc(&d_cdf_g, 256 * sizeof(unsigned char));
    hipMalloc(&d_cdf_b, 256 * sizeof(unsigned char));

    // Copy image data from host to device
    hipMemcpy(d_image, inputImage.data, imageSize, hipMemcpyHostToDevice);

    // Initialize histograms to zero (memset on the device)
    hipMemset(d_hist_r, 0, histSize);
    hipMemset(d_hist_g, 0, histSize);
    hipMemset(d_hist_b, 0, histSize);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Warm-up kernel
    warmUpKernel<<<1, 1>>>();
    CUDA_CHECK(hipDeviceSynchronize());

    // Open the CSV file in append mode
    std::ofstream csvFile("../execution_times_cuda.csv", std::ios::app);
    if (!csvFile.is_open()) {
        std::cerr << "Error: Could not open CSV file!" << std::endl;
        return;
    }

    // Write the header if the file is empty
    if (csvFile.tellp() == 0) {
        csvFile << "Stage,Time (ms),Width,Height,Channels,TileWidth,TileHeight" << std::endl;
    }

    // Record total execution start time
    hipEventRecord(start);

    // Timing the histogram computation kernel
    hipEvent_t stage_start, stage_stop;
    hipEventCreate(&stage_start);
    hipEventCreate(&stage_stop);
    hipEventRecord(stage_start);
    
    // Define block and grid sizes
    dim3 block(tile_width, tile_height);
    dim3 grid((width + tile_width - 1) / tile_width, (height + tile_height - 1) / tile_height);
    // Launch the compute histogram kernel
    computeHistogram<<<grid, block>>>(d_image, d_hist_r, d_hist_g, d_hist_b, width, height);
    hipEventRecord(stage_stop);
    hipDeviceSynchronize();
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, stage_start, stage_stop);
    std::cout << "Histogram computation time " << milliseconds << " ms" << std::endl;
    csvFile << "Histogram computation," << milliseconds << "," << width << "," << height << "," << channels << "," << tile_width << "," << tile_height << std::endl;

    // Timing the CDF computation kernel
    hipEventRecord(stage_start);
    // Compute the CDF for each channel (each one requires its own kernel)
    computeCDF<<<dim3(1, 1), 256>>>(d_hist_r, d_cdf_r, width, height);
    computeCDF<<<dim3(1, 1), 256>>>(d_hist_g, d_cdf_g, width, height);
    computeCDF<<<dim3(1, 1), 256>>>(d_hist_b, d_cdf_b, width, height);
    hipEventRecord(stage_stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&milliseconds, stage_start, stage_stop);
    std::cout << "CDF computation time " << milliseconds << " ms" << std::endl;
    csvFile << "CDF computation," << milliseconds << "," << width << "," << height << "," << channels << "," << tile_width << "," << tile_height << std::endl;

    // Timing the equalization kernel
    hipEventRecord(stage_start);
    // Launch the equalization kernel
    equalizeRGBImage<<<grid, block>>>(d_image, d_output, width, height, d_cdf_r, d_cdf_g, d_cdf_b, tile_width, tile_height);
    hipEventRecord(stage_stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&milliseconds, stage_start, stage_stop);
    std::cout << "Equalization time " << milliseconds << " ms" << std::endl;
    csvFile << "Equalization," << milliseconds << "," << width << "," << height << "," << channels << "," << tile_width << "," << tile_height << std::endl;

    // Record total execution end time and calculate total time
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Total execution time " << milliseconds << " ms" << std::endl;

    // Save the total execution time in the CSV
    csvFile << "Total execution," << milliseconds << "," << width << "," << height << "," << channels << "," << tile_width << "," << tile_height << std::endl;

    // Copy the result back to the host
    hipMemcpy(inputImage.data, d_output, imageSize, hipMemcpyDeviceToHost);

    // Construct the output filename
    std::stringstream outputFileName;
    outputFileName << "../outputs/RGB_output_image_" << width << "x" << height << "_CUDA.jpg";

    // Save the processed image
    if (!cv::imwrite(outputFileName.str(), inputImage)) {
        std::cerr << "Error: Could not save the image!" << std::endl;
    } else {
        std::cout << "Image saved as " << outputFileName.str() << std::endl;
    }

    // Free device memory
    hipFree(d_image);
    hipFree(d_output);
    hipFree(d_hist_r);
    hipFree(d_hist_g);
    hipFree(d_hist_b);
    hipFree(d_cdf_r);
    hipFree(d_cdf_g);
    hipFree(d_cdf_b);

    // Destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(stage_start);
    hipEventDestroy(stage_stop);

    // Close the CSV file
    csvFile.close();
}





int main() {
    // Load the original image as grayscale and color
    cv::Mat inputImageGray = cv::imread("../images/img2.bmp", cv::IMREAD_GRAYSCALE);
    if (inputImageGray.empty()) {
        std::cerr << "Error: Could not load grayscale image!" << std::endl;
        return -1;
    }

    cv::Mat inputImageColor = cv::imread("../images/img2.bmp", cv::IMREAD_COLOR);
    if (inputImageColor.empty()) {
        std::cerr << "Error: Could not load color image!" << std::endl;
        return -1;
    }

    // List of sizes to process (square resolutions)
    std::vector<int> sizes = {128, 256, 512, 1024, 2048};

    // Loop through each resolution
    for (int size : sizes) {
        // Resize images to the current resolution (size x size)
        cv::Mat resizedGray, resizedColor;
        cv::resize(inputImageGray, resizedGray, cv::Size(size, size));
        cv::resize(inputImageColor, resizedColor, cv::Size(size, size));

        // Process grayscale image
        //std::cout << "Processing grayscale image at resolution: " << size << "x" << size << std::endl;
        //equalizeImageWithCUDAGrayscale(resizedGray);

        // Process color image
        std::cout << "Processing color image at resolution: " 
                  << size << "x" << size << std::endl;
        equalizeImageWithCUDA(resizedColor, 16, 16);
    }

    return 0;
}
