#include "UtilityCuda.cu"

#include <fstream>
#include <iostream>

void equalizeImageWithCUDAGrayscale(const cv::Mat& inputImage) {
    int width = inputImage.cols;
    int height = inputImage.rows;
    int channels = 1;  // Since it's a grayscale image, we set channels to 1.
    const int TILE_WIDTH = 16;  // Tile width for shared memory
    const int TILE_HEIGHT = 16; // Tile height for shared memory

    // Device memory allocation
    unsigned char *d_image, *d_output;
    int *d_hist;
    unsigned char *d_cdf;

    // CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // Start overall time measurement
    CUDA_CHECK(hipEventRecord(start));

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_image, width * height * sizeof(unsigned char)));
    CUDA_CHECK(hipMemcpy(d_image, inputImage.data, width * height * sizeof(unsigned char), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_output, width * height * sizeof(unsigned char)));
    CUDA_CHECK(hipMalloc(&d_hist, 256 * sizeof(int)));
    CUDA_CHECK(hipMemset(d_hist, 0, 256 * sizeof(int)));

    CUDA_CHECK(hipMalloc(&d_cdf, 256 * sizeof(unsigned char)));

    // ----------------------------------------
    // Compute Histogram
    // ----------------------------------------

    hipEvent_t histStart, histStop;
    CUDA_CHECK(hipEventCreate(&histStart));
    CUDA_CHECK(hipEventCreate(&histStop));
    CUDA_CHECK(hipEventRecord(histStart));

    // Compute histogram on the device using the updated kernel
    dim3 block(TILE_WIDTH, TILE_HEIGHT);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    computeHistogramGrayscale<<<grid, block>>>(d_image, d_hist, width, height);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(histStop));
    CUDA_CHECK(hipEventSynchronize(histStop));

    float histMilliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&histMilliseconds, histStart, histStop));
    std::cout << "Histogram computation time: " << histMilliseconds << " ms.\n";

    // ----------------------------------------
    // Compute CDF
    // ----------------------------------------

    hipEvent_t cdfStart, cdfStop;
    CUDA_CHECK(hipEventCreate(&cdfStart));
    CUDA_CHECK(hipEventCreate(&cdfStop));
    CUDA_CHECK(hipEventRecord(cdfStart));

    // Compute CDF on the device
    computeCDFGrayscale<<<1, 256>>>(d_hist, d_cdf, width, height);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(cdfStop));
    CUDA_CHECK(hipEventSynchronize(cdfStop));

    float cdfMilliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&cdfMilliseconds, cdfStart, cdfStop));
    std::cout << "CDF computation time: " << cdfMilliseconds << " ms.\n";

    // ----------------------------------------
    // Histogram Equalization
    // ----------------------------------------

    hipEvent_t equalizationStart, equalizationStop;
    CUDA_CHECK(hipEventCreate(&equalizationStart));
    CUDA_CHECK(hipEventCreate(&equalizationStop));
    CUDA_CHECK(hipEventRecord(equalizationStart));

    // Perform histogram equalization using the updated kernel with shared memory tiling
    equalizeGrayscaleImage<<<grid, block>>>(d_image, d_output, width, height, d_cdf);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(equalizationStop));
    CUDA_CHECK(hipEventSynchronize(equalizationStop));

    float equalizationMilliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&equalizationMilliseconds, equalizationStart, equalizationStop));
    std::cout << "Histogram equalization time: " << equalizationMilliseconds << " ms.\n";

    // Stop overall time measurement
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    // Calculate the total elapsed time
    float totalMilliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&totalMilliseconds, start, stop));

    std::cout << "Total Histogram Equalization (Grayscale) executed in " << totalMilliseconds << " ms.\n";

    // Copy result back to host
    cv::Mat outputImage(height, width, CV_8UC1);
    CUDA_CHECK(hipMemcpy(outputImage.data, d_output, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost));

    // Cleanup
    CUDA_CHECK(hipFree(d_image));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_hist));
    CUDA_CHECK(hipFree(d_cdf));

    // Save the output
    cv::imwrite("../outputs/cuda_equalized_grayscale_image.jpg", outputImage);

    // Save the total execution time to the CSV file
    std::ofstream csvFile;
    csvFile.open("execution_times.csv", std::ios::app);

    // Write CSV header if the file is empty
    if (csvFile.tellp() == 0) {
        csvFile << "Stage,Time (ms),Width,Height,Channels,TileWidth,TileHeight" << std::endl;
    }

    // Write the data for each stage
    csvFile << "Histogram Computation," << histMilliseconds << "," 
            << width << "," << height << "," << channels << ","
            << TILE_WIDTH << "," << TILE_HEIGHT << std::endl;

    csvFile << "CDF Computation," << cdfMilliseconds << "," 
            << width << "," << height << "," << channels << ","
            << TILE_WIDTH << "," << TILE_HEIGHT << std::endl;

    csvFile << "Equalization," << equalizationMilliseconds << "," 
            << width << "," << height << "," << channels << ","
            << TILE_WIDTH << "," << TILE_HEIGHT << std::endl;

    csvFile << "Total," << totalMilliseconds << "," 
            << width << "," << height << "," << channels << ","
            << TILE_WIDTH << "," << TILE_HEIGHT << std::endl;

    // Close the CSV file
    csvFile.close();
}






void equalizeImageWithCUDA(const cv::Mat& inputImage, int tile_width, int tile_height) {
    int width = inputImage.cols;
    int height = inputImage.rows;
    int channels = inputImage.channels();  // Get the number of channels

    // Allocate memory for input image and output image
    unsigned char *d_image, *d_output;
    int *d_hist_r, *d_hist_g, *d_hist_b;
    unsigned char *d_cdf_r, *d_cdf_g, *d_cdf_b;

    // Allocate device memory (ensure it's the correct size)
    size_t imageSize = width * height * 3 * sizeof(unsigned char);
    size_t histSize = 256 * sizeof(int);

    hipMalloc(&d_image, imageSize);
    hipMalloc(&d_output, imageSize);
    hipMalloc(&d_hist_r, histSize);
    hipMalloc(&d_hist_g, histSize);
    hipMalloc(&d_hist_b, histSize);
    hipMalloc(&d_cdf_r, 256 * sizeof(unsigned char));
    hipMalloc(&d_cdf_g, 256 * sizeof(unsigned char));
    hipMalloc(&d_cdf_b, 256 * sizeof(unsigned char));

    // Copy image data from host to device
    hipMemcpy(d_image, inputImage.data, imageSize, hipMemcpyHostToDevice);

    // Initialize histograms to zero (memset on the device)
    hipMemset(d_hist_r, 0, histSize);
    hipMemset(d_hist_g, 0, histSize);
    hipMemset(d_hist_b, 0, histSize);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Open the CSV file in append mode
    std::ofstream csvFile("../execution_times_cuda.csv", std::ios::app);
    if (!csvFile.is_open()) {
        std::cerr << "Error: Could not open CSV file!" << std::endl;
        return;
    }

    // Write the header if the file is empty
    if (csvFile.tellp() == 0) {
        csvFile << "Stage,Time (ms),Width,Height,Channels,TileWidth,TileHeight" << std::endl;
    }

    // Record total execution start time
    hipEventRecord(start);

    // Timing the histogram computation kernel
    hipEvent_t stage_start, stage_stop;
    hipEventCreate(&stage_start);
    hipEventCreate(&stage_stop);
    hipEventRecord(stage_start);
    
    // Define block and grid sizes
    dim3 block(tile_width, tile_height);
    dim3 grid((width + tile_width - 1) / tile_width, (height + tile_height - 1) / tile_height);
    // Launch the compute histogram kernel
    computeHistogram<<<grid, block>>>(d_image, d_hist_r, d_hist_g, d_hist_b, width, height);
    hipEventRecord(stage_stop);
    hipDeviceSynchronize();
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, stage_start, stage_stop);
    std::cout << "Histogram computation time " << milliseconds << " ms" << std::endl;
    csvFile << "Histogram computation," << milliseconds << "," << width << "," << height << "," << channels << "," << tile_width << "," << tile_height << std::endl;

    // Timing the CDF computation kernel
    hipEventRecord(stage_start);
    // Compute the CDF for each channel (each one requires its own kernel)
    computeCDF<<<dim3(1, 1), 256>>>(d_hist_r, d_cdf_r, width, height);
    computeCDF<<<dim3(1, 1), 256>>>(d_hist_g, d_cdf_g, width, height);
    computeCDF<<<dim3(1, 1), 256>>>(d_hist_b, d_cdf_b, width, height);
    hipEventRecord(stage_stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&milliseconds, stage_start, stage_stop);
    std::cout << "CDF computation time " << milliseconds << " ms" << std::endl;
    csvFile << "CDF computation," << milliseconds << "," << width << "," << height << "," << channels << "," << tile_width << "," << tile_height << std::endl;

    // Timing the equalization kernel
    hipEventRecord(stage_start);
    // Launch the equalization kernel
    equalizeRGBImage<<<grid, block>>>(d_image, d_output, width, height, d_cdf_r, d_cdf_g, d_cdf_b, tile_width, tile_height);
    hipEventRecord(stage_stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&milliseconds, stage_start, stage_stop);
    std::cout << "Equalization time " << milliseconds << " ms" << std::endl;
    csvFile << "Equalization," << milliseconds << "," << width << "," << height << "," << channels << "," << tile_width << "," << tile_height << std::endl;

    // Record total execution end time and calculate total time
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Total execution time " << milliseconds << " ms" << std::endl;

    // Save the total execution time in the CSV
    csvFile << "Total execution," << milliseconds << "," << width << "," << height << "," << channels << "," << tile_width << "," << tile_height << std::endl;

    // Copy the result back to the host
    hipMemcpy(inputImage.data, d_output, imageSize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_image);
    hipFree(d_output);
    hipFree(d_hist_r);
    hipFree(d_hist_g);
    hipFree(d_hist_b);
    hipFree(d_cdf_r);
    hipFree(d_cdf_g);
    hipFree(d_cdf_b);

    // Destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(stage_start);
    hipEventDestroy(stage_stop);

    // Close the CSV file
    csvFile.close();
}





int main() {
    // Load the original image as grayscale and color
    cv::Mat inputImageGray = cv::imread("../images/img2.bmp", cv::IMREAD_GRAYSCALE);
    if (inputImageGray.empty()) {
        std::cerr << "Error: Could not load grayscale image!" << std::endl;
        return -1;
    }

    cv::Mat inputImageColor = cv::imread("../images/img2.bmp", cv::IMREAD_COLOR);
    if (inputImageColor.empty()) {
        std::cerr << "Error: Could not load color image!" << std::endl;
        return -1;
    }

    // List of sizes to process (square resolutions)
    std::vector<int> sizes = {128};

    // Loop through each resolution
    for (int size : sizes) {
        // Resize images to the current resolution (size x size)
        cv::Mat resizedGray, resizedColor;
        cv::resize(inputImageGray, resizedGray, cv::Size(size, size));
        cv::resize(inputImageColor, resizedColor, cv::Size(size, size));

        // Process grayscale image
        std::cout << "Processing grayscale image at resolution: " 
                  << size << "x" << size << std::endl;
        equalizeImageWithCUDAGrayscale(resizedGray);

        // Process color image
        std::cout << "Processing color image at resolution: " 
                  << size << "x" << size << std::endl;
        equalizeImageWithCUDA(resizedColor, 16, 16);
    }

    return 0;
}
