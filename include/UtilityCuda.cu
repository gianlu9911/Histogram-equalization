#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <fstream>
#include <iomanip>

// CUDA error checking 
#define CUDA_CHECK(call) \
{ \
    const hipError_t error = call; \
    if (error != hipSuccess) \
    { \
        fprintf(stderr, "CUDA Error at %s:%d: %s\n", __FILE__, __LINE__, #call); \
        fprintf(stderr, "Error code: %d, Reason: %s\n", error, hipGetErrorString(error)); \
        exit(EXIT_FAILURE); \
    } \
}

// Histogram Kernel with Shared Memory
__global__ void compute_histogram(const unsigned char* d_input, int* d_hist, int width, int height) {
    __shared__ int hist_shared[256];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (threadIdx.x < 256) hist_shared[threadIdx.x] = 0;
    __syncthreads();

    if (idx < width * height) {
        atomicAdd(&hist_shared[d_input[idx]], 1);
    }
    __syncthreads();

    if (threadIdx.x < 256) {
        atomicAdd(&d_hist[threadIdx.x], hist_shared[threadIdx.x]);
    }
}


__global__ void compute_cdf(int* d_hist, int* d_cdf, int total_pixels) {
    __shared__ int hist_shared[256];
    __shared__ int cdf_shared[256];
    int idx = threadIdx.x;

    if (idx < 256) hist_shared[idx] = d_hist[idx];
    __syncthreads();

    if (idx < 256) {
        cdf_shared[idx] = 0;
        for (int i = 0; i <= idx; ++i) {
            cdf_shared[idx] += hist_shared[i];
        }
        cdf_shared[idx] = (cdf_shared[idx] * 255) / total_pixels;
    }
    __syncthreads();

    if (idx < 256) d_cdf[idx] = cdf_shared[idx];
}

// Equalization Kernel with Shared Memory
__global__ void equalize_image(unsigned char* d_output, const unsigned char* d_input, const int* d_cdf, int width, int height) {
    __shared__ int cdf_shared[256];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (threadIdx.x < 256) {
        cdf_shared[threadIdx.x] = d_cdf[threadIdx.x];
    }
    __syncthreads();

    if (idx < width * height) {
        d_output[idx] = cdf_shared[d_input[idx]];
    }
}
=======
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <utility>

__global__ void compute_histogram(const unsigned char* d_input, int* d_hist, int width, int height) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < width * height) {
        atomicAdd(&d_hist[d_input[idx]], 1);
    }
}

__global__ void compute_cdf(int* d_hist, int* d_cdf, int total_pixels) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 256) {
        d_cdf[idx] = 0;
        for (int i = 0; i <= idx; ++i) {
            d_cdf[idx] += d_hist[i];
        }
        d_cdf[idx] = (d_cdf[idx] * 255) / total_pixels;
    }
}

__global__ void equalize_image(unsigned char* d_output, const unsigned char* d_input, const int* d_cdf, int width, int height) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < width * height) {
        d_output[idx] = d_cdf[d_input[idx]];
    }
}
=======
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <utility>

__global__ void compute_histogram(const unsigned char* d_input, int* d_hist, int width, int height) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < width * height) {
        atomicAdd(&d_hist[d_input[idx]], 1);
    }
}

__global__ void compute_cdf(int* d_hist, int* d_cdf, int total_pixels) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 256) {
        d_cdf[idx] = 0;
        for (int i = 0; i <= idx; ++i) {
            d_cdf[idx] += d_hist[i];
        }
        d_cdf[idx] = (d_cdf[idx] * 255) / total_pixels;
    }
}

__global__ void equalize_image(unsigned char* d_output, const unsigned char* d_input, const int* d_cdf, int width, int height) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < width * height) {
        d_output[idx] = d_cdf[d_input[idx]];
    }
}
>>>>>>> fa413aa0b83c7d85b8316d3827d75df66d921c36
>>>>>>> 82380bd8a704aa1303f781b68e5e46aed0cfa4c8
