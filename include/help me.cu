#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <chrono>
#include "Utility.h"

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {
    // Measure time for histogram equalization
    auto start_hist = std::chrono::high_resolution_clock::now();

    // Defining input file name (input image name)
    std::string input_file_name = "dataset/img1.bmp";

    // Defining output file name (output image name)
    std::string output_file_name = "img.png";

    // Calling function to perform histogram equalization
    my_histogramEqualisation(input_file_name, output_file_name);

    auto end_hist = std::chrono::high_resolution_clock::now();
    auto hist_duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_hist - start_hist).count();
    std::cout << "Histogram Equalization Time: " << hist_duration << " ms" << std::endl;

    // Example vectors for CUDA addition
    const int arraySize = 256;
    int a[arraySize], b[arraySize], c[arraySize];
    for (int i = 0; i < arraySize; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Measure time for CUDA addition
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "CUDA Kernel Execution Time: " << milliseconds << " ms" << std::endl;

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel.
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Wait for the kernel to finish, and check for errors.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}